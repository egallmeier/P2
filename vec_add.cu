
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <math.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i+=stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements
  //std::vector<float> x(N);
  //std::vector<float> y(N);

  float *x, *y;

  //Allocate unified memory- accessible from cpu or gpu

  hipError_t errx = hipMallocManaged(&x, N*sizeof(float));
  hipError_t erry = hipMallocManaged(&y, N*sizeof(float));
  if (errx != hipSuccess || erry != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(errx));
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(erry));
  }
  // initialize x and y arrays on the host
  //for (auto& xi : x) xi = 1.0f;
  //for (auto& yi : y) yi = 2.0f;
 
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize -1) / blockSize;
  // Run kernel on 1M elements on the GPU
  add<<<numBlocks,blockSize>>>(N,x,y);//(N, x.data(), y.data());

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)//(auto yi : y)
    maxError = fmax(maxError, fabs(y[i]-3.0f)); //yi
  std::cout << "Max error: " << maxError << std::endl;
  
  // free memory
  hipFree(x);
  hipFree(y);
  return 0;
}
